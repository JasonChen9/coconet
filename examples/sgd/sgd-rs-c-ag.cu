#include "hip/hip_runtime.h"
#include "header.h"
__global__ void binOpFunc0(int N, float lr, float * g, float * w, float * S1, int comm_size, int rank) {
  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  S1[i0] = (w[DIVUP(N, comm_size) * rank + i0] - (g[DIVUP(N, comm_size) * rank + i0] * lr));
}

void sgd(int N, float lr, float* g, float* w, float* S1, float& elapsedTimebinOpFunc0, float& elapsedTimeAllGather, float& elapsedTimeReduceScatter, ncclComm_t comm, hipStream_t stream, int comm_size, int rank){
  hipEvent_t startsgd, stopsgd;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startsgd));
  CUDACHECK(hipEventCreate(&stopsgd));

  CUDACHECK(hipEventRecord(startsgd, stream));
  ncclReduceScatter(g, g, DIVUP(N, comm_size), ncclFloat32, ncclSum, comm, stream);
  CUDACHECK(hipEventRecord(stopsgd, stream));
  CUDACHECK(hipEventSynchronize(stopsgd));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startsgd,stopsgd));
  elapsedTimeReduceScatter += elapsedTime;

  CUDACHECK(hipEventRecord(startsgd, stream));
  size_t totalThreads_0 = (size_t)DIVUP(N, comm_size);
  size_t numThreads_0 = (size_t)min(totalThreads_0, 256UL);
  size_t numThreadBlocks_0 = DIVUP(totalThreads_0, numThreads_0);
  binOpFunc0<<<numThreadBlocks_0, numThreads_0, 0, stream>>>(N, lr, g, w, S1, comm_size, rank);
  CUDACHECK(hipEventRecord(stopsgd, stream));
  CUDACHECK(hipEventSynchronize(stopsgd));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startsgd,stopsgd));
  elapsedTimebinOpFunc0 += elapsedTime;

  CUDACHECK(hipEventRecord(startsgd, stream));
  ncclAllGather(S1, w, DIVUP(N, comm_size), ncclFloat32, comm, stream);
  CUDACHECK(hipEventRecord(stopsgd, stream));
  CUDACHECK(hipEventSynchronize(stopsgd));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startsgd,stopsgd));
  elapsedTimeAllGather += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  MPI_Barrier(MPI_COMM_WORLD);

  for (int __i = 10; __i < 30; __i++) {
    size_t N = 1 << __i;
    // Inputs
    float* g;
    CUDACHECK(hipMalloc(&g, N * sizeof(float)));
    cudaMemRandInt(g, N);
    float* w;
    CUDACHECK(hipMalloc(&w, N * sizeof(float)));
    cudaMemRandInt(w, N);
    float lr;
    lr = 1.0f;

    // Outputs

    // Intermediates
    float* S1;
    CUDACHECK(hipMalloc(&S1, DIVUP(N, comm_size) * sizeof(float)));
    float elapsedTimebinOpFunc0 = 0;
    float elapsedTimeAllGather = 0;
    float elapsedTimeReduceScatter = 0;
    for(int iter = 0; iter < epochs; iter++) {
      sgd(N, lr, g, w, S1, elapsedTimebinOpFunc0, elapsedTimeAllGather, elapsedTimeReduceScatter, comm, stream, comm_size, rank); 
    }
    CUDACHECK(hipFree(g));
    CUDACHECK(hipFree(w));
    CUDACHECK(hipFree(S1));
    if (rank == 0) 
      printf("{SZ: %ld, Epochs: %d, binOpFunc0: %f, AllGather: %f, ReduceScatter: %f, Total: %f}\n", N, epochs, elapsedTimebinOpFunc0, elapsedTimeAllGather, elapsedTimeReduceScatter, elapsedTimebinOpFunc0 + elapsedTimeAllGather + elapsedTimeReduceScatter);
  }
  MPI_Finalize();
}
