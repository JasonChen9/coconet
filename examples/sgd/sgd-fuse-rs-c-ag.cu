#include "header.h"
void sgd(int N, float lr, float* g, float* w, float& elapsedTimeFusedAllReduce, ncclComm_t comm, hipStream_t stream, int comm_size, int rank){
  hipEvent_t startsgd, stopsgd;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startsgd));
  CUDACHECK(hipEventCreate(&stopsgd));

  CUDACHECK(hipEventRecord(startsgd, stream));
  NCCLCHECK(AllReduce_pipe(lr, beta1, beta2, (half*)g, w, (half*)w, m, v, N, ncclHalf, comm, ncclSum, stream));
  CUDACHECK(hipEventRecord(stopsgd, stream));
  CUDACHECK(hipEventSynchronize(stopsgd));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startsgd,stopsgd));
  elapsedTimeFusedAllReduce += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  MPI_Barrier(MPI_COMM_WORLD);

  for (int __i = 10; __i < 30; __i++) {
    size_t N = 1 << __i;
    // Inputs
    float* g;
    CUDACHECK(hipMalloc(&g, N * sizeof(float)));
    cudaMemRandInt(g, N);
    float* w;
    CUDACHECK(hipMalloc(&w, N * sizeof(float)));
    cudaMemRandInt(w, N);
    float lr;
    lr = 1.0f;

    // Outputs
    float elapsedTimeFusedAllReduce = 0;
    for(int iter = 0; iter < epochs; iter++) {
      sgd(N, lr, g, w, elapsedTimeFusedAllReduce, comm, stream, comm_size, rank); 
    }
    CUDACHECK(hipFree(g));
    CUDACHECK(hipFree(w));
    if (rank == 0) 
      printf("{SZ: %ld, Epochs: %d, FusedAllReduce: %f, Total: %f}\n", N, epochs, elapsedTimeFusedAllReduce, elapsedTimeFusedAllReduce);
  }
  MPI_Finalize();
}
