#include "hip/hip_runtime.h"
#include "header.h"
__global__ void binOpFunc0(int N, float beta1, float beta2, float * g, float * w, float * m, float * v, int comm_size, int rank) {
  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  v[i0] = ((v[i0] * beta2) + ((g[i0] * (1 - beta2)) * g[i0]));
  float S4;
  S4 = (v[i0] / beta2);
  m[i0] = ((m[i0] * beta1) + (g[i0] * (1 - beta1)));
  float S3;
  S3 = (m[i0] / beta1);
  w[i0] = (w[i0] - (S3 / S4));
}

void adam(int N, float lr, float beta1, float beta2, float* g, float* w, float* m, float* v, float& elapsedTimebinOpFunc0, float& elapsedTimeAllReduce, ncclComm_t comm, hipStream_t stream, int comm_size, int rank){
  hipEvent_t startadam, stopadam;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startadam));
  CUDACHECK(hipEventCreate(&stopadam));

  CUDACHECK(hipEventRecord(startadam, stream));
  ncclAllReduce(g, g, N, ncclFloat32,ncclSum, comm, stream);
  CUDACHECK(hipEventRecord(stopadam, stream));
  CUDACHECK(hipEventSynchronize(stopadam));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startadam,stopadam));
  elapsedTimeAllReduce += elapsedTime;

  CUDACHECK(hipEventRecord(startadam, stream));
  size_t totalThreads_0 = (size_t)N;
  size_t numThreads_0 = (size_t)min(totalThreads_0, 256UL);
  size_t numThreadBlocks_0 = DIVUP(totalThreads_0, numThreads_0);
  binOpFunc0<<<numThreadBlocks_0, numThreads_0, 0, stream>>>(N, beta1, beta2, g, w, m, v, comm_size, rank);
  CUDACHECK(hipEventRecord(stopadam, stream));
  CUDACHECK(hipEventSynchronize(stopadam));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startadam,stopadam));
  elapsedTimebinOpFunc0 += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  MPI_Barrier(MPI_COMM_WORLD);

  for (int __i = 10; __i < 30; __i++) {
    size_t N = 1 << __i;
    // Inputs
    float* g;
    CUDACHECK(hipMalloc(&g, N * sizeof(float)));
    cudaMemRandInt(g, N);
    float* w;
    CUDACHECK(hipMalloc(&w, N * sizeof(float)));
    cudaMemRandInt(w, N);
    float* m;
    CUDACHECK(hipMalloc(&m, N * sizeof(float)));
    cudaMemRandInt(m, N);
    float* v;
    CUDACHECK(hipMalloc(&v, N * sizeof(float)));
    cudaMemRandInt(v, N);
    float lr;
    lr = 1.0f;
    float beta1;
    beta1 = 1.0f;
    float beta2;
    beta2 = 1.0f;

    // Outputs
    float elapsedTimebinOpFunc0 = 0;
    float elapsedTimeAllReduce = 0;
    for(int iter = 0; iter < epochs; iter++) {
      adam(N, lr, beta1, beta2, g, w, m, v, elapsedTimebinOpFunc0, elapsedTimeAllReduce, comm, stream, comm_size, rank); 
    }
    CUDACHECK(hipFree(g));
    CUDACHECK(hipFree(w));
    CUDACHECK(hipFree(m));
    CUDACHECK(hipFree(v));
    if (rank == 0) 
      printf("{SZ: %ld, Epochs: %d, binOpFunc0: %f, AllReduce: %f, Total: %f}\n", N, epochs, elapsedTimebinOpFunc0, elapsedTimeAllReduce, elapsedTimebinOpFunc0 + elapsedTimeAllReduce);
  }
  MPI_Finalize();
}
