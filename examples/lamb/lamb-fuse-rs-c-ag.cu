#include "header.h"
void lamb(int N, float lr, float beta1, float beta2, float epsilon, float gamma, float* w, float* g, float* m, float* v, float& elapsedTimeFusedAllReduce, ncclComm_t comm, hipStream_t stream, int comm_size, int rank){
  hipEvent_t startlamb, stoplamb;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startlamb));
  CUDACHECK(hipEventCreate(&stoplamb));

  CUDACHECK(hipEventRecord(startlamb, stream));
  NCCLCHECK(AllReduce_pipe(lr, beta1, beta2, (half*)g, w, (half*)w, m, v, w, N, ncclHalf, comm, ncclSum, stream));
  CUDACHECK(hipEventRecord(stoplamb, stream));
  CUDACHECK(hipEventSynchronize(stoplamb));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startlamb,stoplamb));
  elapsedTimeFusedAllReduce += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  MPI_Barrier(MPI_COMM_WORLD);

  for (int __i = 10; __i < 30; __i++) {
    size_t N = 1 << __i;
    // Inputs
    float* g;
    CUDACHECK(hipMalloc(&g, N * sizeof(float)));
    cudaMemRandInt(g, N);
    float* w;
    CUDACHECK(hipMalloc(&w, N * sizeof(float)));
    cudaMemRandInt(w, N);
    float* m;
    CUDACHECK(hipMalloc(&m, DIVUP(N, comm_size)*comm_size * sizeof(float)));
    cudaMemRandInt(m, DIVUP(N, comm_size));
    float* v;
    CUDACHECK(hipMalloc(&v, DIVUP(N, comm_size)*comm_size * sizeof(float)));
    cudaMemRandInt(v, DIVUP(N, comm_size));
    float lr;
    lr = 1.0f;
    float beta1;
    beta1 = 1.0f;
    float beta2;
    beta2 = 1.0f;
    float epsilon;
    epsilon = 1.0f;
    float gamma;
    gamma = 1.0f;

    // Outputs
    float elapsedTimeFusedAllReduce = 0;
    for(int iter = 0; iter < epochs; iter++) {
      lamb(N, lr, beta1, beta2, epsilon, gamma, w, g, m, v, elapsedTimeFusedAllReduce, comm, stream, comm_size, rank); 
    }
    CUDACHECK(hipFree(g));
    CUDACHECK(hipFree(w));
    CUDACHECK(hipFree(m));
    CUDACHECK(hipFree(v));
    if (rank == 0) 
      printf("{SZ: %ld, Epochs: %d, FusedAllReduce: %f, Total: %f}\n", N, epochs, elapsedTimeFusedAllReduce, elapsedTimeFusedAllReduce);
  }
  MPI_Finalize();
}
