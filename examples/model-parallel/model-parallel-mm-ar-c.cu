#include "hip/hip_runtime.h"
#include "header.h"
#include "cutlass-matmul.h"
void matMul0(int B, int S, int H, half * w, half * in, half * S0, hipblasHandle_t cublasHandle, int comm_size, int rank) {
  const half alpha = __float2half(1.0f);
  const half beta = __float2half(0.0f);
  CUBLASCHECK(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    (H), (B*S), DIVUP((H), comm_size), 
    &alpha,     w, HIP_R_16F, (H), 
    in, HIP_R_16F, DIVUP((H), comm_size), 
    &beta, S0, HIP_R_16F, (H), 
    HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));
}

__global__ void binOpFunc0(int B, int S, int H, half * b, half * S0, half * S2, int comm_size, int rank) {
  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  S2[i0] = (S0[i0] + b[i0%H]);
}

__global__ void binOpFunc1(int B, int S, int H, half * r, half * S0, half * S2, half * S3, int comm_size, int rank) {
  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  hiprandState curandState0;
  hiprand_init(0, 0, 0, &curandState0);
  S3[i0] = ((hiprand_uniform(&curandState0) < 0.5 ? S2[i0] : (half) 0) + r[i0]);
}

void model_parallel(int B, int S, int H, half* w, half* b, half* in, half* r, half* S3, half* S0, half* S2, float& elapsedTimebinOpFunc1, float& elapsedTimebinOpFunc0, float& elapsedTimeAllReduce, float& elapsedTimematMul0, ncclComm_t comm, hipStream_t stream, int comm_size, int rank, hipblasHandle_t cublasHandle){
  hipEvent_t startmodel_parallel, stopmodel_parallel;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startmodel_parallel));
  CUDACHECK(hipEventCreate(&stopmodel_parallel));

  CUDACHECK(hipEventRecord(startmodel_parallel, stream));
  matMul0(B, S, H, w, in, S0, cublasHandle, comm_size, rank);
  CUDACHECK(hipEventRecord(stopmodel_parallel, stream));
  CUDACHECK(hipEventSynchronize(stopmodel_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startmodel_parallel,stopmodel_parallel));
  elapsedTimematMul0 += elapsedTime;

  CUDACHECK(hipEventRecord(startmodel_parallel, stream));
  ncclAllReduce(S0, S0, (B*S*H), ncclHalf,ncclSum, comm, stream);
  CUDACHECK(hipEventRecord(stopmodel_parallel, stream));
  CUDACHECK(hipEventSynchronize(stopmodel_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startmodel_parallel,stopmodel_parallel));
  elapsedTimeAllReduce += elapsedTime;

  CUDACHECK(hipEventRecord(startmodel_parallel, stream));
  size_t totalThreads_1 = (size_t)(B*S*H);
  size_t numThreads_1 = (size_t)min(totalThreads_1, 256UL);
  size_t numThreadBlocks_1 = DIVUP(totalThreads_1, numThreads_1);
  binOpFunc0<<<numThreadBlocks_1, numThreads_1, 0, stream>>>(B, S, H, b, S0, S2, comm_size, rank);
  CUDACHECK(hipEventRecord(stopmodel_parallel, stream));
  CUDACHECK(hipEventSynchronize(stopmodel_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startmodel_parallel,stopmodel_parallel));
  elapsedTimebinOpFunc0 += elapsedTime;

  CUDACHECK(hipEventRecord(startmodel_parallel, stream));
  size_t totalThreads_2 = (size_t)(B*S*H);
  size_t numThreads_2 = (size_t)min(totalThreads_2, 256UL);
  size_t numThreadBlocks_2 = DIVUP(totalThreads_2, numThreads_2);
  binOpFunc1<<<numThreadBlocks_2, numThreads_2, 0, stream>>>(B, S, H, r, S0, S2, S3, comm_size, rank);
  CUDACHECK(hipEventRecord(stopmodel_parallel, stream));
  CUDACHECK(hipEventSynchronize(stopmodel_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startmodel_parallel,stopmodel_parallel));
  elapsedTimebinOpFunc1 += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipblasHandle_t cublasHandle;
  CUBLASCHECK(hipblasCreate(&cublasHandle));
  CUBLASCHECK(hipblasSetStream(cublasHandle, stream));
  CUBLASCHECK(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
  MPI_Barrier(MPI_COMM_WORLD);

  int array_B[] = {8, 16};
  for (int iter_B = 0; iter_B< sizeof(array_B)/sizeof(array_B[0]);iter_B++) {
    int B = array_B[iter_B];
    size_t S = 1024;
    size_t H = 3072;
    // Inputs
    half* w;
    CUDACHECK(hipMalloc(&w, DIVUP((H*H), comm_size) * sizeof(half)));
    cudaMemRandInt(w, DIVUP((H*H), comm_size));
    half* b;
    CUDACHECK(hipMalloc(&b, H * sizeof(half)));
    cudaMemRandInt(b, H);
    half* in;
    CUDACHECK(hipMalloc(&in, DIVUP((B*S*H), comm_size) * sizeof(half)));
    cudaMemRandInt(in, DIVUP((B*S*H), comm_size));
    half* r;
    CUDACHECK(hipMalloc(&r, (B*S*H) * sizeof(half)));
    cudaMemRandInt(r, (B*S*H));

    // Outputs
    half* S3;
    CUDACHECK(hipMalloc(&S3, (B*S*H) * sizeof(half)));

    // Intermediates
    half* S0;
    half* S2;
    CUDACHECK(hipMalloc(&S0, (B*S*H) * sizeof(half)));
    CUDACHECK(hipMalloc(&S2, (B*S*H) * sizeof(half)));
    float elapsedTimebinOpFunc1 = 0;
    float elapsedTimebinOpFunc0 = 0;
    float elapsedTimeAllReduce = 0;
    float elapsedTimematMul0 = 0;
    for(int iter = 0; iter < epochs; iter++) {
      model_parallel(B, S, H, w, b, in, r, S3, S0, S2, elapsedTimebinOpFunc1, elapsedTimebinOpFunc0, elapsedTimeAllReduce, elapsedTimematMul0, comm, stream, comm_size, rank, cublasHandle); 
    }
    CUDACHECK(hipFree(w));
    CUDACHECK(hipFree(b));
    CUDACHECK(hipFree(in));
    CUDACHECK(hipFree(r));
    CUDACHECK(hipFree(S3));
    CUDACHECK(hipFree(S0));
    CUDACHECK(hipFree(S2));
    if (rank == 0) 
      printf("{B: %ld, S: %ld, H: %ld, Epochs: %d, binOpFunc1: %f, binOpFunc0: %f, AllReduce: %f, matMul0: %f, Total: %f}\n", B, S, H, epochs, elapsedTimebinOpFunc1, elapsedTimebinOpFunc0, elapsedTimeAllReduce, elapsedTimematMul0, elapsedTimebinOpFunc1 + elapsedTimebinOpFunc0 + elapsedTimeAllReduce + elapsedTimematMul0);
  }
  MPI_Finalize();
}
