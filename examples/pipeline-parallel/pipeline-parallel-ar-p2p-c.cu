#include "hip/hip_runtime.h"
#include "header.h"
void matMul0(int B, int S, int H, half * w, half * in, half * S0, hipblasHandle_t cublasHandle, int comm_size, int rank) {
  const half alpha = __float2half(1.0f);
  const half beta = __float2half(0.0f);
  CUBLASCHECK(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    (H), (B*S), DIVUP((H), comm_size), 
    &alpha,     w, HIP_R_16F, (H), 
    in, HIP_R_16F, DIVUP((H), comm_size), 
    &beta, S0, HIP_R_16F, (H), 
    HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));
}

__global__ void binOpFunc0(int B, int S, int H, half * r, half * S0, half * S2, int comm_size, int rank) {
  int i0 = threadIdx.x + blockDim.x*blockIdx.x;
  S2[i0] = (S0[i0] + r[i0]);
}

void pipeline_parallel(int B, int S, int H, half* w, half* b, half* in, half* r, half* S2, half* S0, float& elapsedTimebinOpFunc0, float& elapsedTimeAllReduce, float& elapsedTimematMul0, ncclComm_t comm, hipStream_t stream, int comm_size, int rank, hipblasHandle_t cublasHandle){
  hipEvent_t startpipeline_parallel, stoppipeline_parallel;
  float elapsedTime;
  CUDACHECK(hipEventCreate(&startpipeline_parallel));
  CUDACHECK(hipEventCreate(&stoppipeline_parallel));

  CUDACHECK(hipEventRecord(startpipeline_parallel, stream));
  matMul0(B, S, H, w, in, S0, cublasHandle, comm_size, rank);
  CUDACHECK(hipEventRecord(stoppipeline_parallel, stream));
  CUDACHECK(hipEventSynchronize(stoppipeline_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startpipeline_parallel,stoppipeline_parallel));
  elapsedTimematMul0 += elapsedTime;

  CUDACHECK(hipEventRecord(startpipeline_parallel, stream));
  ncclAllReduce(S0, S0, (B*S*H), ncclHalf,ncclSum, comm, stream);
  CUDACHECK(hipEventRecord(stoppipeline_parallel, stream));
  CUDACHECK(hipEventSynchronize(stoppipeline_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startpipeline_parallel,stoppipeline_parallel));
  elapsedTimeAllReduce += elapsedTime;

  CUDACHECK(hipEventRecord(startpipeline_parallel, stream));
  size_t totalThreads_1 = (size_t)(B*S*H);
  size_t numThreads_1 = (size_t)min(totalThreads_1, 256UL);
  size_t numThreadBlocks_1 = DIVUP(totalThreads_1, numThreads_1);
  binOpFunc0<<<numThreadBlocks_1, numThreads_1, 0, stream>>>(B, S, H, r, S0, S2, comm_size, rank);
  CUDACHECK(hipEventRecord(stoppipeline_parallel, stream));
  CUDACHECK(hipEventSynchronize(stoppipeline_parallel));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, startpipeline_parallel,stoppipeline_parallel));
  elapsedTimebinOpFunc0 += elapsedTime;


}
int main(int argc, char** argv){
  //Get number of gpus in the node
  int N_GPUs;
  CUDACHECK(hipGetDeviceCount(&N_GPUs));
  MPI_Init(&argc, &argv);
  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUs));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  ncclCommInitRank(&comm, comm_size, id, rank);
  if (argc < 2) { printf("Specify epochs as command arg"); return 1;}
   int epochs = atoi(argv[1]);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipblasHandle_t cublasHandle;
  CUBLASCHECK(hipblasCreate(&cublasHandle));
  CUBLASCHECK(hipblasSetStream(cublasHandle, stream));
  CUBLASCHECK(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
  MPI_Barrier(MPI_COMM_WORLD);

  int array_B[] = {8, 16};
  for (int iter_B = 0; iter_B< sizeof(array_B)/sizeof(array_B[0]);iter_B++) {
    int B = array_B[iter_B];
    size_t S = 1024;
    size_t H = 3072;
    // Inputs
    half* w;
    CUDACHECK(hipMalloc(&w, DIVUP((H*H), comm_size) * sizeof(half)));
    cudaMemRandInt(w, DIVUP((H*H), comm_size));
    half* b;
    CUDACHECK(hipMalloc(&b, H * sizeof(half)));
    cudaMemRandInt(b, H);
    half* in;
    CUDACHECK(hipMalloc(&in, DIVUP((B*S*H), comm_size) * sizeof(half)));
    cudaMemRandInt(in, DIVUP((B*S*H), comm_size));
    half* r;
    CUDACHECK(hipMalloc(&r, (B*S*H) * sizeof(half)));
    cudaMemRandInt(r, (B*S*H));

    // Outputs
    half* S2;
    CUDACHECK(hipMalloc(&S2, (B*S*H) * sizeof(half)));

    // Intermediates
    half* S0;
    CUDACHECK(hipMalloc(&S0, (B*S*H) * sizeof(half)));
    float elapsedTimebinOpFunc0 = 0;
    float elapsedTimeAllReduce = 0;
    float elapsedTimematMul0 = 0;
    for(int iter = 0; iter < epochs; iter++) {
      pipeline_parallel(B, S, H, w, b, in, r, S2, S0, elapsedTimebinOpFunc0, elapsedTimeAllReduce, elapsedTimematMul0, comm, stream, comm_size, rank, cublasHandle); 
    }
    CUDACHECK(hipFree(w));
    CUDACHECK(hipFree(b));
    CUDACHECK(hipFree(in));
    CUDACHECK(hipFree(r));
    CUDACHECK(hipFree(S2));
    CUDACHECK(hipFree(S0));
    if (rank == 0) 
      printf("{B: %ld, S: %ld, H: %ld, Epochs: %d, binOpFunc0: %f, AllReduce: %f, matMul0: %f, Total: %f}\n", B, S, H, epochs, elapsedTimebinOpFunc0, elapsedTimeAllReduce, elapsedTimematMul0, elapsedTimebinOpFunc0 + elapsedTimeAllReduce + elapsedTimematMul0);
  }
  MPI_Finalize();
}
