#include "hip/hip_runtime.h"
#include "header.h"
void pipe(float lr, float beta1, float beta2, float* g, float* w, float* m, float* v, ncclComm_t comm, hipStream_t stream, size_t SZ){
  NCCLCHECK(AllReduce_pipe(lr, beta1, beta2, g, w, m, v, SZ, ncclFloat32, comm, ncclSum, stream));

  CUDACHECK(hipStreamSynchronize(stream));
}
bool mpiRef(float* __g, float* __w, float* __m, float* __v, float __lr, float __beta1, float __beta2, float* w, float* m, float* v, size_t SZ, bool dummy=false)
{
  float* __S0;
  __S0 = new float[SZ];
  MPI_Allreduce(__g, __S0, SZ, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
  float* __S2;
  __S2 = new float[SZ];
  for (size_t i0 = 0; i0 < SZ; i0++) {
    __S2[i0] = ((__beta2 * __v[i0]) + ((1 - __beta2) * (__S0[i0] * __S0[i0])));
  }
  float* __S4;
  __S4 = new float[SZ];
  for (size_t i0 = 0; i0 < SZ; i0++) {
    __S4[i0] = (__S2[i0] / __beta2);
  }
  float* __S1;
  __S1 = new float[SZ];
  for (size_t i0 = 0; i0 < SZ; i0++) {
    __S1[i0] = ((__beta1 * __m[i0]) + ((1 - __beta1) * __S0[i0]));
  }
  float* __S3;
  __S3 = new float[SZ];
  for (size_t i0 = 0; i0 < SZ; i0++) {
    __S3[i0] = (__S1[i0] / __beta1);
  }
  float* __S5;
  __S5 = new float[SZ];
  float* hS5;
  hS5 = new float[SZ];
  CUDACHECK(hipMemcpy(hS5, w, SZ*sizeof(float), hipMemcpyDeviceToHost));
  for (size_t i0 = 0; i0 < SZ; i0++) {
    __S5[i0] = (__w[i0] - ((__lr * __S3[i0]) / __S4[i0]));
    if (!eqFloat(__S5[i0], hS5[i0])) {
      printf("Mismatch at %ld : ref '%f', computed '%f'\n",i0, __S5[i0], hS5[i0]);
      return false;
    }
  }
  return true;
}
int main(int argc, char** argv){
const int N_GPUS = 16;
    MPI_Init(&argc, &argv);  int comm_size, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  ncclComm_t comm;
  CUDACHECK(hipSetDevice(rank % N_GPUS));
  //initializing NCCL
  ncclUniqueId id;
  if (rank == 0) ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  int epochs = 1010;
  hipStream_t stream;
  hipStreamCreate(&stream);
  if (rank == 0)
    printf("adam\n");
  
    // printf("starting at rank %d for algo %s \n", rank, algo.c_str());
    ncclCommInitRank(&comm, comm_size, id, rank);
    MPI_Barrier(MPI_COMM_WORLD);
    if (rank == 0)
    printf("<results>\n");
    for (int  P = 10; P < 31; P++) {
      size_t SZ = 1UL << P;
      // Inputs
      float* g;
      CUDACHECK(hipMalloc(&g, SZ * sizeof(float)));
      memset_value(g, 1.0f, SZ);
      float* w;
      CUDACHECK(hipMalloc(&w, SZ * sizeof(float)));
      memset_value(w, 0.0f, SZ);
      float* m;
      CUDACHECK(hipMalloc(&m, SZ * sizeof(float)));
      memset_value(m, 0.0f, SZ);
      float* v;
      CUDACHECK(hipMalloc(&v, SZ * sizeof(float)));
      memset_value(v, 0.0f, SZ);
      float lr;
      lr = 1.0f;
      float beta1;
      beta1 = 0.5f;
      float beta2;
      beta2 = 0.5f;

      // Outputs
      float totalTime = 0;
      for(int iter = 0; iter < epochs; iter++) {
        // printf("iter %d\n", iter);
        float* __g;;
        if (iter == 0) {
          __g = new float[SZ];;
          CUDACHECK(hipMemcpy(__g, g, SZ*sizeof(float), hipMemcpyDeviceToHost));;
        }
        float* __w;;
        if (iter == 0) {
          __w = new float[SZ];;
          CUDACHECK(hipMemcpy(__w, w, SZ*sizeof(float), hipMemcpyDeviceToHost));;
        }
        float* __m;;
        if(iter == 0) {
          __m = new float[SZ];;
          CUDACHECK(hipMemcpy(__m, m, SZ*sizeof(float), hipMemcpyDeviceToHost));;
        }
        float* __v;;
        if(iter == 0) {
          __v = new float[SZ];;
          CUDACHECK(hipMemcpy(__v, v, SZ*sizeof(float), hipMemcpyDeviceToHost));;
        }
        float __lr;;
        if (iter == 0)
        __lr = __half2float(lr);
        float __beta1;;
        if(iter == 0)
        __beta1 = __half2float(beta1);
        float __beta2;;
        if (iter == 0)
        __beta2 = __half2float(beta2);
        hipEvent_t startpipe, stoppipe;
        float elapsedTimepipe;
        CUDACHECK(hipEventCreate(&startpipe));
        CUDACHECK(hipEventCreate(&stoppipe));
        CUDACHECK(hipEventRecord(startpipe, 0));
        pipe(lr, beta1, beta2, g, w, m, v, comm, stream,SZ); 
        CUDACHECK(hipEventRecord(stoppipe, 0));
        CUDACHECK(hipEventSynchronize(stoppipe));
        CUDACHECK(hipEventElapsedTime(&elapsedTimepipe, startpipe,stoppipe));
        if (iter > 10)
        totalTime += elapsedTimepipe;
        if (iter == 0) assert(mpiRef(__g, __w, __m, __v, __lr, __beta1, __beta2, w, m, v, false));
      }
      MPI_Barrier(MPI_COMM_WORLD);
      if (rank == 0)
      printf("%s %ld %f ms\n", getenv ("NCCL_ALGO"), SZ, totalTime);

      hipFree(g);
      hipFree(w);
      hipFree(m);
      hipFree(v);
    }
    if(rank == 0)
    printf("</results>\n");
    ncclCommDestroy(comm);

  MPI_Finalize();
}
